#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// turing.cpp : Defines the entry point for the console application.
//

#include <iostream>
#include <vector>
#include <sstream> 
#include <hash_map>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


class MyClass_Hasher {
    public:
     static const size_t bucket_size = 10; // mean bucket size that the container should try not to exceed
     static const size_t min_buckets = (1 << 10); // minimum number of buckets, power 
     MyClass_Hasher() {
          // should be default-constructible
     }
     size_t operator()(const std::vector<int>& v) const {
             size_t hash = 5381;

            std::vector<int>::const_iterator iter = v.begin();
            std::vector<int>::const_iterator end  = v.end();
            for(; iter != end; ++iter) {
                hash = ((hash << 5) + hash) + *iter;
            }

            return hash;
     }

     bool operator()(const std::vector<int>& v1, const std::vector<int>& v2) const {
           if (v1.size() != v2.size()) {
                return true;
            }
            std::vector<int>::const_iterator iter1 = v1.begin();
            std::vector<int>::const_iterator iter2 = v2.begin();
            std::vector<int>::const_iterator end1 = v1.end();
            for (; iter1 != end1; ++iter1, ++iter2) {
                if (*iter1 != *iter2) {
                    return true;
                }
            }

            return false;
     }
 };

   

class Machine 
{
public:

    Machine(long long machineNumber)
        :transitionsTable_(),
        currentStateIndex_(0),
        tape_(),
        currentIndexInTape_(0),
        iterationNumber_(0),
        machineNumber_(machineNumber),
        maxIndexWriteInTape_(0)
    {
        long long nb = machineNumber;
        transitionsTable_.resize(8);

        {
            // inverse order!!!!!
            std::vector<std::vector<int> >::iterator iter = transitionsTable_.begin();
            std::vector<std::vector<int> >::iterator end = transitionsTable_.end();
            for (; iter != end; ++iter) {
                int rest = nb % 16;

                iter->resize(3);
                // inverse order !!!
                std::vector<int>::iterator subIter = iter->begin();
                std::vector<int>::iterator subEnd = iter->end();
                int subNb = rest;
                for (; subIter != subEnd-1; ++subIter) {
                    int subRest = subNb % 2;
                    *subIter = subRest;

                    subNb = (subNb - subRest)/2;
                }
                *subIter = subNb;


                nb = (nb - rest)/16;
            }

        }

        tape_.resize(300, 0);            

    }

    void process(std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >& map)
    {
        for (int iterationCounter = 0; iterationCounter < 300; ++iterationCounter) {
            int charRead = tape_[currentIndexInTape_];

            std::vector<int>* pointerOnInstructionToFollow = NULL;
            if (currentStateIndex_ == 0) {
                if (charRead == 0) {
                    pointerOnInstructionToFollow = &transitionsTable_[0];
                }
                else {
                    pointerOnInstructionToFollow = &transitionsTable_[1];
                }
            }
            else if (currentStateIndex_ == 1) {
                if (charRead == 0) {
                    pointerOnInstructionToFollow = &transitionsTable_[2];
                }
                else {
                    pointerOnInstructionToFollow = &transitionsTable_[3];
                }
            }
            else if (currentStateIndex_ == 2) {
                if (charRead == 0) {
                    pointerOnInstructionToFollow = &transitionsTable_[4];
                }
                else {
                    pointerOnInstructionToFollow = &transitionsTable_[5];
                }
            }
            else {
                if (charRead == 0) {
                    pointerOnInstructionToFollow = &transitionsTable_[6];
                }
                else {
                    pointerOnInstructionToFollow = &transitionsTable_[7];
                }
            }

            // write on the tape
            tape_[currentIndexInTape_] = (*pointerOnInstructionToFollow)[1];

            // move that in the if... see if it give the same.
            

            // Update the index in tape
            if ((*pointerOnInstructionToFollow)[0] == 0) {
                currentIndexInTape_++;
                if (currentIndexInTape_ > maxIndexWriteInTape_) {
                    maxIndexWriteInTape_ = currentIndexInTape_;
                }
            }
            else {
                currentIndexInTape_--;
                if (currentIndexInTape_ < 0) {
                    
                    std::vector<int> infosToStore;
                    infosToStore.reserve(maxIndexWriteInTape_);

                    if (maxIndexWriteInTape_ >= 0) {
                        for (int i = maxIndexWriteInTape_; i > 0; --i) {
                            infosToStore.push_back(tape_[i]);
                        }
                        infosToStore.push_back(tape_[0]);
                    }

                    std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator mapIter = map.find(infosToStore);
                    if (mapIter == map.end()) {
                        map[infosToStore] = std::make_pair(machineNumber_, (iterationCounter+1));
                    }
                    else {
                        if (machineNumber_ < mapIter->second.first) {
                            map[infosToStore] = std::make_pair(machineNumber_, (iterationCounter+1));
                        }
                    }

                    return;
                }
            }

            // Update state to go to.
            currentStateIndex_ = (*pointerOnInstructionToFollow)[2];
        }

    }


    long long machineNumber_;

    int currentStateIndex_;
	
    int currentIndexInTape_;

    int iterationNumber_;

    std::vector<int> tape_;

    int maxIndexWriteInTape_;

    std::vector<std::vector<int> > transitionsTable_; 

};


// La fonction qui tourne sur la carte graphique en parall�le
__global__ void
run_turing_machine(const long long machineNumBegin, bool* machineEnded, int* nbIteration, int* printedTape, int* sizeTape, int tapeLength, int numElements)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= numElements) {
        return;
    }

    // Calculer le num�ro de la machine courante.
    long long machineNumber = machineNumBegin + index;

    // Rejeter les machines impaires.
    if (machineNumber%2 == 1) {
        sizeTape[index] = -1;  // pour forcer le traitement ult�rieur de cette machine.
        machineEnded[index] = true;
        return;
    }
    

	// currentTapepointer est un pointeur qui se met au d�but de la bande allou�e pour la machine courante
    int* currentTapepointer = printedTape + index*tapeLength;
    for (int* tmp = currentTapepointer; tmp  != currentTapepointer+tapeLength; ++tmp) {
		// On initialise toute la bande � 0.
        *tmp = 0;
    }


    int currentStateIndex_ = 0;
    int currentIndexInTape_ = 0;
    int iterationNumber_ = 0;
    int maxIndexWriteInTape_ = 0;
    machineEnded[index] = false;
    sizeTape[index] = 0;

    
    int transitionsTable[3*8];
    {
        int rest;
        int subNb;
        for (int i = 0; i < 8; ++i) {
            rest = machineNumber % 16;

            // inverse order !!!
            subNb = rest;
            int j = 0;
            for (; j < 2; ) {
                int subRest = subNb % 2;
                transitionsTable[3*i+j] = subRest;

                subNb = (subNb - subRest)/2;

                j++;
            }
            transitionsTable[3*i+j] = subNb;

            machineNumber = (machineNumber - rest)/16;
        }
    }

    

    for (int iterationCounter = 0; iterationCounter < 300; ++iterationCounter) {
        int charRead = currentTapepointer[currentIndexInTape_];

        int* pointerOnInstructionToFollow = NULL;
        if (currentStateIndex_ == 0) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*0];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*1];
            }
        }
        else if (currentStateIndex_ == 1) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*2];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*3];
            }
        }
        else if (currentStateIndex_ == 2) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*4];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*5];
            }
        }
        else {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*6];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*7];
            }
        }
        
        // write on the tape
        currentTapepointer[currentIndexInTape_] = pointerOnInstructionToFollow[1];
		

        // Update the index in tape
        if (pointerOnInstructionToFollow[0] == 0) {
            currentIndexInTape_++;
            if (currentIndexInTape_ > maxIndexWriteInTape_) {
                maxIndexWriteInTape_ = currentIndexInTape_;
            }

            if (currentIndexInTape_ >= tapeLength) {
                return;
            }
        }
        else {
            currentIndexInTape_--;
            if (currentIndexInTape_ < 0) {
                machineEnded[index] = true;
                nbIteration[index] = iterationCounter+1;
                sizeTape[index] = maxIndexWriteInTape_;
                return;
            }
        }

        // Update state to go to.
        currentStateIndex_ = pointerOnInstructionToFollow[2];
    }
	    
}



/*bool sym4 (long long int nb)
	{
	long long int quo, nx, min; 
	short i,j,nb16[6][8]={};

	quo=min=nb;
	
	for (i=7; quo>0; i--) {nb16[0][i] = quo%16; quo=quo/16;}

	nb16[1][0]=nb16[2][2]=nb16[3][4]=nb16[4][2]=nb16[5][4]=nb16[0][0];
	nb16[1][1]=nb16[2][3]=nb16[3][5]=nb16[4][3]=nb16[5][5]=nb16[0][1];
	nb16[1][4]=nb16[2][0]=nb16[3][0]=nb16[4][4]=nb16[5][2]=nb16[0][2];
	nb16[1][5]=nb16[2][1]=nb16[3][1]=nb16[4][5]=nb16[5][3]=nb16[0][3];
	nb16[1][2]=nb16[2][4]=nb16[3][2]=nb16[4][0]=nb16[5][0]=nb16[0][4];
	nb16[1][3]=nb16[2][5]=nb16[3][3]=nb16[4][1]=nb16[5][1]=nb16[0][5];
	nb16[1][6]=nb16[2][6]=nb16[3][6]=nb16[4][6]=nb16[5][6]=nb16[0][6];
	nb16[1][7]=nb16[2][7]=nb16[3][7]=nb16[4][7]=nb16[5][7]=nb16[0][7];

	
	for (i=0; i<8; i++) 
	{
		switch (nb16[1][i]/4)
	{case 0 : break;
	 case 1 : nb16[1][i]=nb16[1][i]+4; break;
	 case 2 : nb16[1][i]=nb16[1][i]-4; break;
	 case 3 : break;}

		switch (nb16[2][i]/4)
	{case 0 : break;
	 case 1 : break;
	 case 2 : nb16[2][i]=nb16[2][i]+4; break;
	 case 3 : nb16[2][i]=nb16[2][i]-4; break;}
	
		switch (nb16[3][i]/4)
	{case 0 : break;
	 case 1 : nb16[3][i]=nb16[3][i]+4; break;
	 case 2 : nb16[3][i]=nb16[3][i]+4; break;
	 case 3 : nb16[3][i]=nb16[3][i]-8; break;}

		switch (nb16[4][i]/4)
	{case 0 : break;
	 case 1 : nb16[4][i]=nb16[4][i]+8; break;
	 case 2 : nb16[4][i]=nb16[4][i]-4; break;
	 case 3 : nb16[4][i]=nb16[4][i]-4; break;}
		
		switch (nb16[5][i]/4)
	{case 0 : break;
	 case 1 : nb16[5][i]=nb16[5][i]+8; break;
	 case 2 : break;
	 case 3 : nb16[5][i]=nb16[5][i]-8; break;}
	
	}
	
	for (j=1; j<6; j++) {nx=0;for (i=0; i<8; i++) {nx=16*nx + nb16[j][i];} if (nx<min) min=nx; }
	if (nb == min) return (false); else return (true);
}*/







int main(int argc, char* argv[])
{

    std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >  map;

    // Ce nombre repr�sente le nombre de machines qui vont tourner en parrall�le avant chaque resynchronisation sur le cpu.
    // A priori plus il est grand plus rapide ce sera mais il faut veiller � ne pas d�passer une certaine limite sous peine de ne 
    // plus avoir de m�moire et que le programme plante.
    // Une marge d'erreur est pr�f�rable d'autant plus si le pc fait tourner d'autres choses en m�me temps.
    const int maxSimultaneously = 1024*100;

    // longueur de la bande d'impression relative � chaque machine de turing.
    const int tapeLength = 20;

	// Allocation des structures sur le processeur.
    int* nbIteration = (int*)malloc(maxSimultaneously*sizeof(int));
    bool* machineEnded = (bool*)malloc(maxSimultaneously*sizeof(bool));
    int* tapeSize = (int*) malloc(maxSimultaneously*sizeof(int));
    int* printedTape = (int*)malloc(tapeLength*maxSimultaneously*sizeof(int));
        
    int* d_nbIteration = NULL;
    bool* d_machineEnded = NULL;
    int* d_tapeSize = NULL;
    int* d_printedTape = NULL;
	// Allocation des structures sur le GPU.
    hipMalloc((void **)&d_nbIteration, maxSimultaneously*sizeof(int));
    hipMalloc((void **)&d_machineEnded, maxSimultaneously*sizeof(bool));
    hipMalloc((void **)&d_tapeSize, maxSimultaneously*sizeof(int));
    hipMalloc((void **)&d_printedTape, tapeLength*maxSimultaneously*sizeof(int));

    long long numberMachinesStopped = 0;

	// Remarque : Pour effectuer des petits tests, on peut r�duire le nombre de machine � calculer....
    long long limit = 4294967296;
    for (long long currentMachineNum = 0; currentMachineNum < limit; ) {/*if (sym4(currentMachineNum)) continue;*/

        int maxNbelements = maxSimultaneously;
        if (currentMachineNum+maxSimultaneously > limit) {
            long long tmp = limit - currentMachineNum;
            maxNbelements = (int) tmp;
        }

		// threadsPerBlock et blocksPerGrid sont 2 param�tres demand� par cuda: Ils peuvent �tre modifi� d�pendant
		// de la carte graphique.
        int threadsPerBlock = 1024;
        int blocksPerGrid =maxSimultaneously/threadsPerBlock;//(maxSimultaneously + threadsPerBlock - 1) / threadsPerBlock;
        run_turing_machine<<<blocksPerGrid, threadsPerBlock>>>(currentMachineNum, d_machineEnded, d_nbIteration, d_printedTape, d_tapeSize, tapeLength, maxNbelements);
        
        hipError_t err = hipGetLastError();

        if (err != hipSuccess)
        {
			// Erreur Cuda, peut �tre les param�tres ont des valeurs trop �lev�e pour la carte graphique utilis�e...
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
		// Rappatrier les donn�es de la carte graphique dans les structures processeur afin de pouvoir les traiter.
        hipMemcpy(nbIteration, d_nbIteration, maxSimultaneously*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(machineEnded, d_machineEnded, maxSimultaneously*sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(tapeSize, d_tapeSize, maxSimultaneously*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(printedTape, d_printedTape, tapeLength*maxSimultaneously*sizeof(int*), hipMemcpyDeviceToHost);
        
		// Parcour de tous les �l�ments.
        for (int k = 0; k < maxNbelements; k++) {
			
			// Test la pertinence des donn�es pour la machine k (machine s'est arr�t�e et � �crit quelque chose?)
            if (machineEnded[k] == true && tapeSize[k] >= 0) {
                std::vector<int> infosToStore;
                infosToStore.reserve(tapeSize[k]+1);

				// Mettre l'impression de la bande dans un vecteur afin de pouvoir faire une recherche dans la table de hash.
                int* pointerBegin = &printedTape[k*tapeLength];
                int* pointerEnd = &printedTape[k*tapeLength]+tapeSize[k]+1;
                for (;pointerBegin != pointerEnd; ++pointerBegin) {
                    infosToStore.push_back(*pointerBegin);
                }

                long long curMachine = currentMachineNum+k;
                std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator mapIter = map.find(infosToStore);
                if (mapIter == map.end()) {
					// R�sulat non trouv�, on l'ins�re donc.
                    map[infosToStore] = std::make_pair(curMachine, nbIteration[k]);
                }
                else {
					// R�sultat d�j� existant, on ins�re seulement si le num�ro de la machine est plus petit que celui d�j� stock�.
                    if (curMachine < mapIter->second.first) {
                        map[infosToStore] = std::make_pair(curMachine, nbIteration[k]);
                    }
                }

                numberMachinesStopped++;
            }
        }

        currentMachineNum += maxSimultaneously;
        
    }

	// Lib�ration de la m�moire sur GPU
    hipFree(d_nbIteration);
    hipFree(d_machineEnded);
    hipFree(d_tapeSize);
    hipFree(d_printedTape);

	// Lib�ration de la m�moire sur CPU
    free(nbIteration);
    free(machineEnded);
    free(tapeSize);
    free(printedTape);


	// Voir explication projet 3statesCpu...

    FILE* f = fopen("MT4_Cuda1.txt", "w");

    fprintf(f, "Nombre de machines executees: %lld\n", limit);
    fprintf(f, "Nombre de machines qui se sont arretee: %lld\n", numberMachinesStopped);
    
    std::vector<std::pair<long long, std::pair<int, std::vector<int> > > > allMachines;

    {
        std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator iter = map.begin();
        std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator end = map.end();
        for (; iter != end; ++iter) {
            allMachines.push_back(std::make_pair(iter->second.first, std::make_pair(iter->second.second, iter->first)));
        }
    }

    std::sort(allMachines.begin(), allMachines.end());

    {
        std::vector<std::pair<long long, std::pair<int, std::vector<int> > > >::const_iterator iter = allMachines.begin();
        std::vector<std::pair<long long, std::pair<int, std::vector<int> > > >::const_iterator end  = allMachines.end();
        for (; iter != end; ++iter) {
            const std::vector<int>& vect = (iter->second.second);
            fprintf(f, "{%lld, %d, {", iter->first, iter->second.first);

            if (vect.size() >= 0) {
                std::vector<int>::const_reverse_iterator tapeIter = vect.rbegin();
                std::vector<int>::const_reverse_iterator tapeEnd = vect.rend();
                for (; tapeIter != tapeEnd-1; ++tapeIter) {
                    fprintf(f, "%d, ", *tapeIter);
                }
                fprintf(f, "%d", *tapeIter);
            }

            fprintf(f, "}}\n");
        }        
    }

    fclose(f);
    std::cout << "process terminated" << std::endl;
    std::cin;
}




