#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// turing.cpp : Defines the entry point for the console application.
//

#include <iostream>
#include <vector>
#include <sstream> 
#include <hash_map>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


class MyClass_Hasher {
    public:
     static const size_t bucket_size = 10; // mean bucket size that the container should try not to exceed
     static const size_t min_buckets = (1 << 10); // minimum number of buckets, power 
     MyClass_Hasher() {
          // should be default-constructible
     }
     size_t operator()(const std::vector<int>& v) const {
             size_t hash = 5381;

            std::vector<int>::const_iterator iter = v.begin();
            std::vector<int>::const_iterator end  = v.end();
            for(; iter != end; ++iter) {
                hash = ((hash << 5) + hash) + *iter;
            }

            return hash;
     }

     bool operator()(const std::vector<int>& v1, const std::vector<int>& v2) const {
           if (v1.size() != v2.size()) {
                return true;
            }
            std::vector<int>::const_iterator iter1 = v1.begin();
            std::vector<int>::const_iterator iter2 = v2.begin();
            std::vector<int>::const_iterator end1 = v1.end();
            for (; iter1 != end1; ++iter1, ++iter2) {
                if (*iter1 != *iter2) {
                    return true;
                }
            }

            return false;
     }
 };
   


__global__ void
run_turing_machine(const long long machineNumBegin, bool* machineEnded, int* nbIteration, int* printedTape, int* sizeTape, int tapeLength, int numElements)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= numElements) {
        return;
    }
    if (index%2 == 0) {
        return;
    }

    int* currentTapepointer = printedTape + index*tapeLength;
    for (int* tmp = currentTapepointer; tmp  != currentTapepointer+tapeLength; ++tmp) {
        *tmp = 0;
    }

        
    long long machineNumber = machineNumBegin + index;
    int currentStateIndex_ = 0;
    int currentIndexInTape_ = 0;
    int iterationNumber_ = 0;
    int maxIndexWriteInTape_ = 0;
    machineEnded[index] = false;
    sizeTape[index] = 0;

    
    int transitionsTable[3*10];
    {
        int rest;
        int subNb;
        for (int i = 0; i < 10; ++i) {
            rest = machineNumber % 20;

            // inverse order !!!
            subNb = rest;
            int j = 0;
            for (; j < 2; ) {
                int subRest = subNb % 2;
                transitionsTable[3*i+j] = subRest;

                subNb = (subNb - subRest)/2;

                j++;
            }
            transitionsTable[3*i+j] = subNb;

            machineNumber = (machineNumber - rest)/20;
        }
    }

    

    for (int iterationCounter = 0; iterationCounter < 1000; ++iterationCounter) {
        int charRead = currentTapepointer[currentIndexInTape_];

        int* pointerOnInstructionToFollow = NULL;
        if (currentStateIndex_ == 0) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*0];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*1];
            }
        }
        else if (currentStateIndex_ == 1) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*2];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*3];
            }
        }
        else if (currentStateIndex_ == 2) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*4];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*5];
            }
        }
        else if (currentStateIndex_ == 3) {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*6];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*7];
            }
        }
        else {
            if (charRead == 0) {
                pointerOnInstructionToFollow = &transitionsTable[3*8];
            }
            else {
                pointerOnInstructionToFollow = &transitionsTable[3*9];
            }
        }
        
        // write on the tape
        currentTapepointer[currentIndexInTape_] = pointerOnInstructionToFollow[1];

        // move that in the if... see if it give the same.

        // Update the index in tape
        if (pointerOnInstructionToFollow[0] == 0) {
            currentIndexInTape_++;
            if (currentIndexInTape_ > maxIndexWriteInTape_) {
                maxIndexWriteInTape_ = currentIndexInTape_;
            }

            if (currentIndexInTape_ >= tapeLength) {
                return;
            }
        }
        else {
            currentIndexInTape_--;
            if (currentIndexInTape_ < 0) {
                machineEnded[index] = true;
                nbIteration[index] = iterationCounter+1;
                sizeTape[index] = maxIndexWriteInTape_;
                return;
            }
        }

        // Update state to go to.
        currentStateIndex_ = pointerOnInstructionToFollow[2];
    }

//end:
    
}



int main(int argc, char* argv[])
{

    std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >  map;

    const int maxSimultaneously = 1024*100;
    const int tapeLength = 200;

    int* nbIteration = (int*)malloc(maxSimultaneously*sizeof(int));
    bool* machineEnded = (bool*)malloc(maxSimultaneously*sizeof(bool));
    int* tapeSize = (int*) malloc(maxSimultaneously*sizeof(int));
    int* printedTape = (int*)malloc(tapeLength*maxSimultaneously*sizeof(int));
        
    int* d_nbIteration = NULL;
    bool* d_machineEnded = NULL;
    int* d_tapeSize = NULL;
    int* d_printedTape = NULL;
    hipMalloc((void **)&d_nbIteration, maxSimultaneously*sizeof(int));
    hipMalloc((void **)&d_machineEnded, maxSimultaneously*sizeof(bool));
    hipMalloc((void **)&d_tapeSize, maxSimultaneously*sizeof(int));
    hipMalloc((void **)&d_printedTape, tapeLength*maxSimultaneously*sizeof(int));


    long long limit = 4294967;//2;//96;//90000000;//;
    for (long long currentMachineNum = 0; currentMachineNum < limit; /*++i*/) {

        

        int maxNbelements = maxSimultaneously;
        if (currentMachineNum+maxSimultaneously > limit) {
            long long tmp = limit - currentMachineNum;
            maxNbelements = (int) tmp;
        }


        const int threadsPerBlock = 1024;
        int blocksPerGrid = maxSimultaneously/threadsPerBlock;//(maxSimultaneously + threadsPerBlock - 1) / threadsPerBlock;
        //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        run_turing_machine<<<blocksPerGrid, threadsPerBlock>>>(currentMachineNum, d_machineEnded, d_nbIteration, d_printedTape, d_tapeSize, tapeLength, maxNbelements);
        //long long test = 67686362;
        //run_turing_machine<<<, threadsPerBlock>>>(test, d_machineEnded, d_nbIteration, d_printedTape, d_tapeSize, tapeLength, maxNbelements);

        hipError_t err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        hipMemcpy(nbIteration, d_nbIteration, maxSimultaneously*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(machineEnded, d_machineEnded, maxSimultaneously*sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(tapeSize, d_tapeSize, maxSimultaneously*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(printedTape, d_printedTape, tapeLength*maxSimultaneously*sizeof(int*), hipMemcpyDeviceToHost);
        
        for (int k = 0; k < maxNbelements; k++) {
            if (machineEnded[k] == true && tapeSize[k] >= 0) {
                std::vector<int> infosToStore;
                infosToStore.reserve(tapeSize[k]+1);

                int* pointerBegin = &printedTape[k*tapeLength];
                int* pointerEnd = &printedTape[k*tapeLength]+tapeSize[k]+1;
                for (;pointerBegin != pointerEnd; ++pointerBegin) {
                    infosToStore.push_back(*pointerBegin);
                }

                long long curMachine = currentMachineNum+k;
                std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator mapIter = map.find(infosToStore);
                if (mapIter == map.end()) {
                    map[infosToStore] = std::make_pair(curMachine, nbIteration[k]);
                }
                else {
                    if (curMachine < mapIter->second.first) {
                        map[infosToStore] = std::make_pair(curMachine, nbIteration[k]);
                    }
                }
            }
        }

        currentMachineNum += maxSimultaneously;
        
    }

    hipFree(d_nbIteration);
    hipFree(d_machineEnded);
    hipFree(d_tapeSize);
    hipFree(d_printedTape);

    free(nbIteration);
    free(machineEnded);
    free(tapeSize);
    free(printedTape);

    FILE* f = fopen("MT5.txt", "w");
    
    std::vector<std::pair<long long, std::pair<int, std::vector<int> > > > allMachines;

    {
        std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator iter = map.begin();
        std::hash_map <std::vector<int>, std::pair<long long, int>, MyClass_Hasher >::const_iterator end = map.end();
        for (; iter != end; ++iter) {
            allMachines.push_back(std::make_pair(iter->second.first, std::make_pair(iter->second.second, iter->first)));
        }
    }

    std::sort(allMachines.begin(), allMachines.end());

    {
        std::vector<std::pair<long long, std::pair<int, std::vector<int> > > >::const_iterator iter = allMachines.begin();
        std::vector<std::pair<long long, std::pair<int, std::vector<int> > > >::const_iterator end  = allMachines.end();
        for (; iter != end; ++iter) {
            const std::vector<int>& vect = (iter->second.second);
            fprintf(f, "{%lld, %d, {", iter->first, iter->second.first);

            if (vect.size() >= 0) {
                std::vector<int>::const_reverse_iterator tapeIter = vect.rbegin();
                std::vector<int>::const_reverse_iterator tapeEnd = vect.rend();
                for (; tapeIter != tapeEnd-1; ++tapeIter) {
                    fprintf(f, "%d, ", *tapeIter);
                }
                fprintf(f, "%d", *tapeIter);
            }

            fprintf(f, "}}\n");
        }        
    }

    fclose(f);
    std::cout << "process terminated" << std::endl;
    std::cin;
}




